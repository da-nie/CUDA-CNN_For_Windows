#include "cuda.cu.h"

#include <cstdio>
#include <ctime>
#include <stdio.h>
#include <stdint.h>

#include "handle_error.cu.h"
#include "ccudacnn.cu.h"

//----------------------------------------------------------------------------------------------------
//главная функция программы на CUDA
//----------------------------------------------------------------------------------------------------
void CUDA_Start(void)
{
 int deviceCount;
 hipDeviceProp_t devProp;

 HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
 printf("Found %d devices\n",deviceCount);
 for(int device=0;device<deviceCount;device++)
 {
  HANDLE_ERROR(hipGetDeviceProperties(&devProp,device));
  printf("Device %d\n", device );
  printf("Compute capability     : %d.%d\n",devProp.major,devProp.minor);
  printf("Name                   : %s\n",devProp.name);
  printf("Total Global Memory    : %d\n",devProp.totalGlobalMem);
  printf("Shared memory per block: %d\n",devProp.sharedMemPerBlock);
  printf("Registers per block    : %d\n",devProp.regsPerBlock);
  printf("Warp size              : %d\n",devProp.warpSize);
  printf("Max threads per block  : %d\n",devProp.maxThreadsPerBlock);
  printf("Total constant memory  : %d\n",devProp.totalConstMem);
 }
 HANDLE_ERROR(hipSetDevice(0));
 HANDLE_ERROR(hipDeviceReset());

 CCUDACNN<float> cCUDACNN;
 cCUDACNN.Execute();
}

